
#include <hip/hip_runtime.h>
#include <stdint.h>

struct ATTableTy {
  uintptr_t HstPtrBegin;
  uintptr_t HstPtrEnd;
  uintptr_t TgtPtrBegin;
};

extern "C" { // disable name mangling

// Translate function
// Binary search version
__device__ void *AddrTrans(void* addr, struct ATTableTy* table) {
    int size = table[0].HstPtrBegin;
    uintptr_t ret = 0;
    uintptr_t addr_int = (intptr_t) addr;
    int head = 1, end = size + 1;
    while (head < end) {
        int mid = (head + end) >> 1;
        if (addr_int >= table[mid].HstPtrBegin) {
            if (addr_int < table[mid].HstPtrEnd) {
                ret = addr_int - table[mid].HstPtrBegin + table[mid].TgtPtrBegin;
                break;
            }
            head = mid+1;
        } else {
            end = mid;
        }
    }
    // Don't fault when notfound
    if (ret == 0) {
        for (int i = 1; i <= size; i++) {
            if (addr_int >= table[i].HstPtrBegin && addr_int < table[i].HstPtrEnd) {
                ret = addr_int - table[i].HstPtrBegin + table[i].TgtPtrBegin;
                break;
            }
        }
    }
    if (ret == 0) {
        return addr;
    }
    return (void*)ret;
}

// Only id 0 do this
__device__ struct ATTableTy *StoreTableShared(struct ATTableTy* table, struct ATTableTy *sm,
        int8_t size /* max size of table in sm */, int32_t tid) {
    int table_size = table[0].HstPtrBegin + 1;
    // if oversize
    if (table_size > size) {
        return table;
    }
    if (tid != 0) {
        return sm;
    }
    // memcpy
    for (int i = 0; i < table_size; i++) {
        sm[i] = table[i];
    }
    return sm;
}
// sync to wait at return
}
