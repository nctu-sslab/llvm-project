
#include <hip/hip_runtime.h>
#include <stdio.h>

void *mymalloc_uvm(size_t size) {
    void *ret;
    hipMallocManaged ((hipDeviceptr_t*)&ret, size, hipMemAttachGlobal);

    return ret;
}

void myfree_uvm(void *ptr) {
    hipFree((hipDeviceptr_t)ptr);
}

void *myrealloc_uvm(void *ptr, size_t size) {

    //cuMemAllocManaged ((CUdeviceptr*)&ret, size, );
    return NULL;
}
int main() {
    void *p = mymalloc_uvm(10);
    printf("%p\n", p);
    return 0;
}
