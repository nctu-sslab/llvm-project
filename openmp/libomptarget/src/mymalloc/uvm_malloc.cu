#include <stdio.h>
#include "mmprivate.h"

void *mymalloc_uvm(size_t size) {
    void *ret = NULL;
    hipError_t err = hipMallocManaged (
            (hipDeviceptr_t*)&ret, size, hipMemAttachGlobal);
    if (err != hipSuccess) {
      puts("Error hipMallocManaged\n");
      //CUDA_ERR_STRING(err);
      const char *errStr;
      hipDrvGetErrorString(err, &errStr);
      printf("%s",errStr);
      return NULL;
    }
    return ret;
}

void myfree_uvm(void *ptr) {
    hipFree((hipDeviceptr_t)ptr);
}

void *myrealloc_uvm(void *ptr, size_t size) {

    //hipMallocManaged ((hipDeviceptr_t*)&ret, size, );
    return NULL;
}
