
#include <hip/hip_runtime.h>
#include <stdint.h>

struct ATTableTy {
  uintptr_t HstPtrBegin;
  uintptr_t HstPtrEnd;
  uintptr_t TgtPtrBegin;
};

extern "C" { // disable name mangling

// Translate function
// Binary search version
#define SM_TABLE_SIZE 20
__shared__ struct ATTableTy sm_table[SM_TABLE_SIZE];
__device__ static struct ATTableTy *tableptr;

__device__ static int flag = 0;

__device__ void *AddrTransTable(void* addr) {
    int size = tableptr[0].HstPtrBegin;
    uintptr_t ret = 0;
    uintptr_t addr_int = (intptr_t) addr;
    int head = 1, end = size;
    int mid;
    while (head <= end) {
        mid = (head + end) >> 1;
            // TODO don't check end to increase perf
        if (addr_int >= tableptr[mid].HstPtrBegin) {
            if (addr_int < tableptr[mid].HstPtrEnd) {
                ret = addr_int - tableptr[mid].HstPtrBegin + tableptr[mid].TgtPtrBegin;
                break;
            } else {
                end = mid -1;
            }
        } else {
            head = mid+1;
        }
    }
    if (ret == 0) {
        printf("fall back addrtrans: %p mid=%d end=%d, size:%d\n",addr, mid, end,size);
        return addr;
    }
    return (void*)ret;
}

// Only id 0 of the block does this
__device__ struct ATTableTy *StoreTableShared(struct ATTableTy* table) {
    int32_t id = threadIdx.x; // warning missing blockdim
    size_t table_size;
    if (id != 0) {
        goto end;
    }
    // FIXME
        tableptr = table;
        goto end;
    table_size = table[0].HstPtrBegin + 1;
    // if oversize
    if (table_size > SM_TABLE_SIZE) {
        tableptr = table;
        goto end;
    }
    // memcpy TODO cuda has memcpy
    tableptr = sm_table;
    for (int i = 0; i < table_size; i++) {
        tableptr[i] = table[i];
    }
end:
    // sync to wait at return
    __syncthreads();
    return tableptr;
}

__device__ void *AddrTransOffset(void *addr, intptr_t *offsets) {
    intptr_t mask = offsets[0];
    intptr_t shift = offsets[1];
    int index = ((intptr_t)addr & mask) >> shift;
    uintptr_t ret = ((uintptr_t)addr + offsets[index+2]);
    return (void*)ret;
}

#define DEFAULT_CM_ENTRY 16
__constant__ intptr_t ConstMem[DEFAULT_CM_ENTRY];
__device__ void *AddrTransOffset2(void *addr, intptr_t *offsets) {
    int id = threadIdx.x;
    intptr_t mask = ConstMem[0];
    intptr_t shift = ConstMem[1];
    int index = ((intptr_t)addr & mask) >> shift;
    uintptr_t ret = ((uintptr_t)addr + ConstMem[index+2]);
    return (void*)ret;
}
/*
__device__ void *AddrTransMask(void *addr) {

    int32_t id = threadIdx.x; // warning missing blockdim
    uintptr_t ret = ((uintptr_t)addr + 0x0000200000000000L);
    //printf("%d: %p->%p\n",id, addr,(void*)ret);
    return (void*)ret;
}
*/

/*
// TODO QQ God bless
__device__ void ConcurrentTrans(int count) {
    int32_t id = threadIdx.x; // warning missing blockdim
    //sm[idx] = AddrTrans(sm[idx]);
    for (start: end) {
    }
}*/
}
